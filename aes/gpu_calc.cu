#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "calculation.h"


void device_sub_bytes(int *state, int *d_sbox) {
  int i, j;
  unsigned char *cb=(unsigned char*)state;
  for(i=0; i<NBb; i+=4){
    for(j=0; j<4; j++){
      cb[i+j] = d_sbox[cb[i+j]];
    }
  }
}

void device_shift_rows(int *state) {
  int i, j, i4;
  unsigned char *cb = (unsigned char*)state;
  unsigned char cw[NBb];
  memcpy(cw, cb, sizeof(cw));

  for (i = 0;i < NB; i+=4) {
    i4 = i*4;
    for(j = 1; j < 4; j++){
      cw[i4+j+0*4] = cb[i4+j+((j+0)&3)*4];
      cw[i4+j+1*4] = cb[i4+j+((j+1)&3)*4];
      cw[i4+j+2*4] = cb[i4+j+((j+2)&3)*4];
      cw[i4+j+3*4] = cb[i4+j+((j+3)&3)*4];
    }
  }
  memcpy(cb,cw,sizeof(cw));
}

void device_mix_columns(int *state) {
  int i, i4, x;
  for(i = 0; i< NB; i++){
    i4 = i*4;
    x  =  mul(dataget(state,i4+0),2) ^
          mul(dataget(state,i4+1),3) ^
          mul(dataget(state,i4+2),1) ^
          mul(dataget(state,i4+3),1);
    x |= (mul(dataget(state,i4+1),2) ^
          mul(dataget(state,i4+2),3) ^
          mul(dataget(state,i4+3),1) ^
          mul(dataget(state,i4+0),1)) << 8;
    x |= (mul(dataget(state,i4+2),2) ^
          mul(dataget(state,i4+3),3) ^
          mul(dataget(state,i4+0),1) ^
          mul(dataget(state,i4+1),1)) << 16;
    x |= (mul(dataget(state,i4+3),2) ^
          mul(dataget(state,i4+0),3) ^
          mul(dataget(state,i4+1),1) ^
          mul(dataget(state,i4+2),1)) << 24;
    state[i] = x;
  }
}

void device_add_round_key(int *state, int *w, int n)
{
  int i;
  for (i = 0; i <NB; i++) {
    state[i] ^= w[i + NB * n];
  }
}

__global__ void device_aes_encrypt(unsigned char *pt, int *rkey,
    unsigned char *ct, const int *d_sbox, long int size) {

  //This kernel executes AES encryption on a GPU.
  //Please modify this kernel!!
  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;

  if(thread_id == 0)
    printf("size = %ld\n", size);

  printf("You can use printf function to eliminate bugs in your kernel.\n");
  printf("This thread ID is %d.\n", thread_id);

  int rnd;
  int data[NB];
  memcpy(data , pt + 16 * thread_id, NBb);

  device_add_round_key(data, rkey, 0);

  for (rnd = 1; rnd < NR; rnd++) {
    device_sub_bytes(data, d_sbox);
    device_shift_rows(data);
    device_mix_columns(data, rkey);
    device_add_round_key(data, rkey, rnd);
  }

  device_sub_bytes(data, d_sbox);
  device_shift_rows(data);
  device_add_round_key(data, rkey, rnd);

  memcpy(ct + 16 * thread_id , data, NBb);
}

void launch_aes_kernel(unsigned char *pt, int *rk, unsigned char *ct, long int size){

  //This function launches the AES kernel.
  //Please modify this function for AES kernel.
  //In this function, you need to allocate the device memory and so on.

  unsigned char *d_pt, *d_ct;
  int *d_rkey;
  int *d_sbox;

  dim3 dim_grid(1,1,1), dim_block(1,1,1);

  hipMalloc((void **)&d_pt, sizeof(unsigned char)*size);
  hipMalloc((void **)&d_rkey, sizeof(int)*44);
  hipMalloc((void **)&d_ct, sizeof(unsigned char)*size);
  hipMalloc((void **)&d_sbox, sizeof(int) * 256);

  hipMemset(d_pt, 0, sizeof(unsigned char)*size);
  hipMemcpy(d_pt, pt, sizeof(unsigned char)*size, hipMemcpyHostToDevice);
  hipMemcpy(d_rkey, rk, sizeof(int)*44, hipMemcpyHostToDevice);
  hipMemcpy(d_sbox, Sbox, sizeof(int) * 256, hipMemcpyHostToDevice);

  device_aes_encrypt<<<dim_grid, dim_block>>>(d_pt, d_rkey, d_ct, d_sbox, size);
  hipMemcpy(ct, d_ct, sizeof(unsigned char)*size, hipMemcpyDeviceToHost);

  hipFree(d_sbox);
  hipFree(d_pt);
  hipFree(d_rkey);
  hipFree(d_ct);
}












